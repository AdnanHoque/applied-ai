#include "hip/hip_runtime.h"

#include <pybind11/pybind11.h>
#include "stochastic_rounding.hpp"
#include <random>

namespace py = pybind11;

__host__ int getOptimalBlockSize() {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    return std::min(prop.maxThreadsPerBlock, 256);
}

torch::Tensor stochastic_round_bf16_cuda(torch::Tensor input, bool requires_grad) {
    TORCH_CHECK(input.is_cuda(), "Input tensor must be on CUDA device");
    TORCH_CHECK(input.is_contiguous(), "Input tensor must be contiguous");
    TORCH_CHECK(input.scalar_type() == torch::kFloat32, "Input tensor must be float32");

    const int threads_per_block = 256;
    const int num_elements = input.numel();
    const int elements_per_thread = 4;

    const int min_blocks = (num_elements + elements_per_thread * threads_per_block - 1) /
                          (elements_per_thread * threads_per_block);

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    const int blocks_per_sm = 4;
    const int min_blocks_for_sms = prop.multiProcessorCount * blocks_per_sm;
    const int num_blocks = std::max(min_blocks, min_blocks_for_sms);

    auto options = torch::TensorOptions()
                      .dtype(torch::kBFloat16)
                      .device(input.device())
                      .requires_grad(requires_grad);
    auto output = torch::empty_like(input, options);

    std::random_device rd;
    std::mt19937_64 gen(rd());
    std::uniform_int_distribution<unsigned long long> dis;
    const unsigned long long seed = dis(gen);

    stochastic_round_bf16<<<num_blocks, threads_per_block>>>(
        input.data_ptr<float>(),
        reinterpret_cast<__hip_bfloat16*>(output.data_ptr()),
        num_elements,
        seed);

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess,
                "CUDA kernel execution failed: ", hipGetErrorString(err));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("stochastic_round_bf16",
          static_cast<torch::Tensor (*)(torch::Tensor, bool)>(&stochastic_round_bf16_cuda),
          "Stochastic rounding to BFloat16",
          py::arg("input"),
          py::arg("requires_grad") = false);
}
